
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

using namespace std;

__global__ void VectorsPairMaximums(size_t size, double *first, double *second, double *res) {
	size_t begin = (size_t) (blockDim.x * blockIdx.x + threadIdx.x);
	size_t offset = gridDim.x * blockDim.x;

	for (size_t i = begin; i < size; i += offset) {
		res[i] = max(first[i], second[i]);
	}
}

__host__ int main(void) {
	size_t size;
	cin >> size;

	double *first = new double[size];
	double *second = new double[size];
	double *res = new double[size];

	for (size_t i = 0; i < size; i++) {
		cin >> first[i];
		//first[i] = i;
	}
	for (size_t i = 0; i < size; i++) {
		cin >> second[i];
		//second[i] = i;
	}

	double *cudaFirst;
	double *cudaSecond;
	double *cudaRes;

	hipMalloc((void**) &cudaFirst, sizeof(double) * size);
	hipMalloc((void**) &cudaSecond, sizeof(double) * size);
	hipMalloc((void**) &cudaRes, sizeof(double) * size);

	hipMemcpy(cudaFirst, first, sizeof(double) * size, hipMemcpyHostToDevice);
	hipMemcpy(cudaSecond, second, sizeof(double) * size, hipMemcpyHostToDevice);

	VectorsPairMaximums<<<256, 256>>>(size, cudaFirst, cudaSecond, cudaRes);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(res, cudaRes, sizeof(double) * size, hipMemcpyDeviceToHost);

	//double *testArr = new double[size];
	//cudaMemcpy(testArr, cudaFirst, sizeof(double) * size, cudaMemcpyDeviceToHost);

	hipEventDestroy(syncEvent);
	hipFree(cudaFirst);
	hipFree(cudaSecond);
	hipFree(cudaRes);

	for (size_t i = 0; i < size; i++) {
		if (i > 0) {
			cout << " ";
		}
		cout << scientific << res[i];
	}
	cout << endl;

	delete [] first;
	delete [] second;
	delete [] res;

	return 0;
}