
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

using namespace std;

__global__ void VectorsPairMaximums(double *first, double *second, double *res) {
	size_t current = (size_t) threadIdx.x;

	res[current] = max(first[current], second[current]);
}

__host__ int main(void) {
	size_t size;
	cin >> size;

	double *first = new double[size];
	double *second = new double[size];
	double *res = new double[size];

	for (size_t i = 0; i < size; i++) {
		cin >> first[i];
	}
	for (size_t i = 0; i < size; i++) {
		cin >> second[i];
	}

	double *cudaFirst;
	double *cudaSecond;
	double *cudaRes;

	hipMalloc((void**) &cudaFirst, sizeof(double) * size);
	hipMalloc((void**) &cudaSecond, sizeof(double) * size);
	hipMalloc((void**) &cudaRes, sizeof(double) * size);

	hipMemcpy(cudaFirst, first, sizeof(double) * size, hipMemcpyHostToDevice);
	hipMemcpy(cudaSecond, second, sizeof(double) * size, hipMemcpyHostToDevice);

	VectorsPairMaximums<<<1, size>>>(cudaFirst, cudaSecond, cudaRes);

	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(res, cudaRes, sizeof(double) * size, hipMemcpyDeviceToHost);

	hipEventDestroy(syncEvent);
	hipFree(cudaFirst);
	hipFree(cudaSecond);
	hipFree(cudaRes);

	for (size_t i = 0; i < size; i++) {
		if (i > 0) {
			cout << " ";
		}
		cout << scientific << res[i];
	}
	cout << endl;

	delete [] first;
	delete [] second;
	delete [] res;

	return 0;
}